#include "hip/hip_runtime.h"
#include <torch/extension.h>

#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>

namespace cuda_ext {


__global__ void mul_kernel(int numel, const float* a, const float* b, float* result) {
  int idx = blockIdx.x * blockDim.x + threadIdx.x;
  if (idx < numel) result[idx] = a[idx] * b[idx];
}

at::Tensor mymul_cuda(const at::Tensor& a, const at::Tensor& b) {
  TORCH_CHECK(a.sizes() == b.sizes());
  TORCH_CHECK(a.dtype() == at::kFloat);
  TORCH_CHECK(b.dtype() == at::kFloat);
  TORCH_INTERNAL_ASSERT(a.device().type() == at::DeviceType::CUDA);
  TORCH_INTERNAL_ASSERT(b.device().type() == at::DeviceType::CUDA);
  at::Tensor a_contig = a.contiguous();
  at::Tensor b_contig = b.contiguous();
  at::Tensor result = torch::empty(a_contig.sizes(), a_contig.options());
  const float* a_ptr = a_contig.data_ptr<float>();
  const float* b_ptr = b_contig.data_ptr<float>();
  float* result_ptr = result.data_ptr<float>();
  int numel = a_contig.numel();
  mul_kernel<<<(numel+255)/256, 256>>>(numel, a_ptr, b_ptr, result_ptr);
  return result;
}


// Registers CUDA implementations for mymul
TORCH_LIBRARY_IMPL(cuda_ext, CUDA, m) {
  m.impl("mymul", &mymul_cuda);
}

}